#include "hip/hip_runtime.h"
/*
 * paper.cu
 * Main program file for PAPER
 *   - loads files from disk using inputModule 
 *   - optimizes overlap with deviceOverlay
 *   - outputs transformation matrices for max overlap
 *  
 * Author: Imran Haque, 2010
 * Copyright 2009-2010, Stanford University
 *
 * This file is licensed under the terms of the GPL. Please see
 * the COPYING file in the accompanying source distribution for
 * full license terms.
 *
 */
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include "inputModule.h"
#include "cudaVolumeTypes.h"
#include "hostAnalyticVolume.h"
#include "deviceAnalyticVolume.h"
#include "deviceOverlay.h"
#include "transformTools.h"
#include <sys/time.h>

#define ELTS7(x) x[0],x[1],x[2],x[3],x[4],x[5],x[6]

double getustime(void) { // {{{
    struct timeval tv;
    gettimeofday(&tv,NULL);
    double t = tv.tv_sec*1e6 + tv.tv_usec;
    return t;
} //}}}

int main(int argc,char** argv) 
{ // {{{
	if (argc < 3) {
		printf("paper [GPU ID] [listing file] \n");
        printf("or\n");
		printf("paper [GPU ID] [reference sdf] [fit sdf] [[fit sdf] ...] \n");
		return 1;
	}
    const int gpuID = atoi(argv[1]);
    hipSetDevice(gpuID);
    fprintf(stderr,"# Executing on GPU %d\n",gpuID);


    //printf("\n++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++\n");

    // Load reference and fit molecules from disk into CUDAmols and dCUDAMultimols {{{
    CUDAmol refmol;
    CUDAmol* fitmols;
    float3 com_ref,*com_fit;
    dCUDAMultimol hostRefMM,hostFitMM,devRefMM,devFitMM;
    uint* molids;
    uint totalMols,distinctMols;
    float* transforms;
    size_t transform_pitch;
    loadMolecules(argc-1,argv+1,
                  &fitmols,refmol,&molids,&transforms,transform_pitch,
                  hostFitMM,devFitMM,
                  hostRefMM,devRefMM,
                  com_ref,&com_fit,
                  totalMols,distinctMols);
    uint nfitmols = totalMols;
    
    //printf("Loaded %d distinct fit molecules, with %d total fit molecules\n",distinctMols,totalMols);
    //printf("Ref molecule: %d atoms\n",refmol.natoms);
    //uint lastid=totalMols+1;
    //for (uint i = 0; i < totalMols; i++) {
        //if (molids[i] != lastid)
        //    printf("Fit molecule %d (id = %d): %d atoms\n",i,molids[i],fitmols[i].natoms);
        //printf("\tTransform = [%f,%f,%f,%f,%f,%f,%f]\n",ELTS7((hostFitMM.transforms+i*hostFitMM.transform_pitch)));
        //float* matrix = transformToCompensatedMatrix(hostFitMM.transforms+i*hostFitMM.transform_pitch,com_ref,com_fit[i]);
        //printTransformMatrix(matrix);
        //free(matrix);
        //lastid=molids[i];
    //}

    /*for (uint i = 0; i < hostFitMM.maxatoms; i++) {
        printf("atom[%d] = [%f,%f,%f,%f]\n",i,hostFitMM.mols[i],hostFitMM.mols[i+hostFitMM.pitch],hostFitMM.mols[i+2*hostFitMM.pitch],hostFitMM.mols[i+3*hostFitMM.pitch]);
    }*/
    //}}}

    
    float* hostDeviceOverlaps = (float*)malloc(nfitmols*sizeof(float));

    // Allocate space for numTimers on-GPU timers per molecule
    const int numTimers = 8;
    bool timingActivated = false;
    clock_t* hostTimings = (clock_t*)malloc(numTimers*sizeof(clock_t));

    float* hostDeviceTransforms = (float*)malloc(nfitmols*devFitMM.transform_pitch*sizeof(float));


    // optimize the overlaps
    double optstart = getustime();
    const int itercount = optimize_sepkernels(devFitMM,devRefMM,hostDeviceOverlaps,hostTimings,numTimers,com_ref,com_fit);
    double optend = getustime();

    fprintf(stderr,"# Optimization used %d iterations of BFGS\n",itercount);

    // Get the transforms back
    hipMemcpy(hostDeviceTransforms,devFitMM.transforms,nfitmols*transform_pitch*sizeof(float),hipMemcpyDeviceToHost);
    /*printf("\nAfter optimization:\n"); 
    for (int i = 0 ; i < nfitmols; i++) {
        float *xf = hostDeviceTransforms+i*transform_pitch;
        float *matrix = transformToCompensatedMatrix(hostDeviceTransforms+i*transform_pitch,com_ref,com_fit[i]);
        printf("[ %.2f %.2f %.2f; %.2f %.2f %.2f %.2f]\n",xf[0],xf[1],xf[2],xf[3],xf[4],xf[5],xf[6]);

        printf("%f\n",hostDeviceOverlaps[i]);
        printTransformMatrix(matrix,stdout);
        free(matrix);
    }*/
    //printf("\n");

    // Max-reduce over starting positions for each molecule
    float* bestOverlaps = new float[distinctMols];
    float* bestTransforms = new float[distinctMols*7];
    memset(bestOverlaps,0,distinctMols*sizeof(float));
    for (uint i = 0; i < totalMols; i++) {
        uint molid = molids[i];
        if (hostDeviceOverlaps[i] > bestOverlaps[molid]) {
            bestOverlaps[molid] = hostDeviceOverlaps[i];
            memcpy(bestTransforms+molid*7,hostDeviceTransforms+i*transform_pitch,7*sizeof(float));
        }
    }
    if (timingActivated) {
        printf("Size of clock_t on host side is %d\n",(int)sizeof(clock_t));
        for (int i = 0; i < numTimers; i++) {
            printf("Timer %d: %lld\n",i,(long long)(hostTimings[i]));
        }
        printf("Average clocks per operation:\n");
        printf("   Line-search: %f\n",(double)(hostTimings[0])/hostTimings[1]);
        printf("   Objective: %f * %f = %f\n",(double)(hostTimings[2])/hostTimings[3],(double)(hostTimings[3])/hostTimings[1],(double)(hostTimings[2])/hostTimings[1]);
        printf("   BFGS update: %f\n",(double)(hostTimings[4])/hostTimings[5]);
        printf("   Gradient: %f\n",(double)(hostTimings[6])/hostTimings[7]);
    }
            

    bool showresults = true;
    bool benchmark   = false;
    if (showresults) { //{{{
        for (uint i = 0; i < nfitmols; i++) {
            /*CUDAmol finalmol;
            transformCUDAmolToHost(fitmols[i],finalmol,hostDeviceTransforms+i*transform_pitch);
            float hostOvl = hostOverlapVolume(refmol,finalmol);
            delete[] finalmol.atoms;
            printf("Final transform: [%f,%f,%f , %f,%f,%f,%f]\n",hostDeviceTransforms[i*transform_pitch],hostDeviceTransforms[i*transform_pitch+1],hostDeviceTransforms[i*transform_pitch+2],hostDeviceTransforms[i*transform_pitch+3],hostDeviceTransforms[i*transform_pitch+4],hostDeviceTransforms[i*transform_pitch+5],hostDeviceTransforms[i*transform_pitch+6]);
            printf("Fit molecule %d: starting overlap: %f, (putative) ending overlap: %f, ending device overlap: %f, ending host overlap: %f\n\n",i,hostDeviceStartOvl[i],hostDeviceOverlaps[i],hostDeviceEndOvl[i],hostOvl);*/
        }
        for (uint i = 0; i < distinctMols; i++) {
            //printf("Molecule id #%d: optimal overlap value = %f\n",i,bestOverlaps[i]);
            //float *xf = bestTransforms+i*7;
            //printf("[ %.2f %.2f %.2f; %.2f %.2f %.2f %.2f]\n",xf[0],xf[1],xf[2],xf[3],xf[4],xf[5],xf[6]);
            float* matrix = transformToCompensatedMatrix(bestTransforms+i*7,com_ref,com_fit[i]);
            printTransformMatrix(matrix,stdout);
            free(matrix);
        }
        //printf("Optimization kernel took %f ms (%f ms/mol)\n",(optend-optstart)/1000.0,(optend-optstart)/(1000.0*distinctMols));
    } //}}}
    if (benchmark) { //{{{
        uint bench_runs = 10;
        double start = getustime();
        // Benchmarking iteration: copy transforms to device, execute kernel, bring back transforms and overlaps
        for (uint i = 0; i < bench_runs; i++) {
            // Copy refmol over
            hipMemcpy(devRefMM.mols,hostRefMM.mols,4*hostRefMM.nmols*devRefMM.pitch*sizeof(float),hipMemcpyHostToDevice);
            hipMemcpy(devRefMM.atomcounts,hostRefMM.atomcounts,1*sizeof(uint),hipMemcpyHostToDevice);
            
            // Copy fitmols and transforms over
            hipMemcpy(devFitMM.mols,hostFitMM.mols,4*hostFitMM.nmols*devFitMM.pitch*sizeof(float),hipMemcpyHostToDevice);
            hipMemcpy(devFitMM.atomcounts,hostFitMM.atomcounts,hostFitMM.nmols*sizeof(uint),hipMemcpyHostToDevice);
            hipMemcpy(devFitMM.molids,hostFitMM.molids,hostFitMM.nmols*sizeof(uint),hipMemcpyHostToDevice);
            hipMemcpy(devFitMM.transforms,hostFitMM.transforms,nfitmols*transform_pitch*sizeof(float),hipMemcpyHostToDevice);
            // Run optimization
            const int itercount = optimize_sepkernels(devFitMM,devRefMM,hostDeviceOverlaps,hostTimings,numTimers,com_ref,com_fit);
            
            // Copy results back
            hipMemcpy(hostDeviceTransforms,devFitMM.transforms,nfitmols*transform_pitch*sizeof(float),hipMemcpyDeviceToHost);
            //hipMemcpy(hostDeviceOverlaps,deviceOverlaps,nfitmols*sizeof(float),hipMemcpyDeviceToHost);
            hipDeviceSynchronize();
            memset(bestOverlaps,0,distinctMols*sizeof(float));
            for (uint i = 0; i < totalMols; i++) {
                uint molid = molids[i];
                if (hostDeviceOverlaps[i] > bestOverlaps[molid]) {
                    bestOverlaps[molid] = hostDeviceOverlaps[i];
                    memcpy(bestTransforms+molid*7,hostDeviceTransforms+i*transform_pitch,7*sizeof(float));
                }
            }
        }
        double end = getustime();
        double runtime = ((end-start)/1000)/bench_runs;
        printf("Benchmark results over %d iterations on %d molecules (%d mol/starts): %f ms/batch optimization, %f ms/molecule, %f ms/position\n",bench_runs,distinctMols,totalMols,runtime,runtime/distinctMols,runtime/totalMols);

    } //}}}
	
    delete[] bestOverlaps;
    delete[] bestTransforms;
	return 0;
} //}}}
